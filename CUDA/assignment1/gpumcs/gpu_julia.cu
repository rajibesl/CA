#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__device__ inline int iterate_pixel(float x, float y, float c_re, float c_im)
{
	int c=0; 
	float z_re=x;
	float z_im=y;
	while (c<255) {
		float re2=z_re*z_re;
		float im2=z_im*z_im;
		if ((re2+im2) > 4) break; 
		z_im=2*z_re*z_im + c_im;
		z_re=re2-im2 + c_re;
		c++;
	}
	return c;
}

__global__ void calc_fractal(int width, int height, float c_re, float c_im, unsigned char* dest)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	float f_x=(float)(x*0.8f)/(float)(width)-0.8f;
	float f_y=(float)(y*0.8f)/(float)(height)-0.8f;

	dest[x+y*width]=iterate_pixel(f_x,f_y,c_re,c_im);
}
//bullshit

// Write a width by height 8-bit color image into File "filename"
void write_ppm(unsigned char* data,unsigned int width,unsigned int height,char* filename)
{
	if (data == NULL) {
		printf("Provide a valid data pointer!\n");
		return;
	}
	if (filename == NULL) {
		printf("Provide a valid filename!\n");
		return;
	}
	if ( (width>4096) || (height>4096)) {
		printf("Only pictures upto 4096x4096 are supported!\n");
		return;
	}
	FILE *f=fopen(filename,"wb");
	if (f == NULL) 
	{
		printf("Opening File %s failed!\n",filename);
		return;
	}
	if (fprintf(f,"P6 %i %i 255\n",width,height) <= 0) {
		printf("Writing to file failed!\n");
		return;
	};
	int i;
	for (i=0;i<height;i++) {
		unsigned char buffer[4096*3];
		int j;
		for (j=0;j<width;j++) {
			int v=data[i*width+j];
			int s;
			s= v << 0;
			s=s > 255? 255 : s;
			buffer[j*3+0]=s;
			s= v << 1;
			s=s > 255? 255 : s;
			buffer[j*3+1]=s;
			s= v << 2;
			s=s > 255? 255 : s;
			buffer[j*3+2]=s;
		}
		if (fwrite(buffer,width*3,1,f) != 1) {
			printf("Writing of line %i to file failed!\n",i);
			return;
		}
	}
	fclose(f);
}


int main(int argc, char** argv) {
	int blockx = atoi(argv[1]);
	int blocky = atoi(argv[2]);

	int imagesize = 256*256;
	unsigned char* image=(unsigned char*)malloc(imagesize);
	assert(image != NULL);

	unsigned char *in;
	hipMalloc((void**)&in,sizeof(unsigned char)*imagesize);

	dim3 block(blockx, blocky); 
	dim3 grid(256/block.x, 256/block.y); 
		
	//calc_fractal(256,256,0.28,0.008,image);
	calc_fractal<<<grid,block>>>(256,256,0.28,0.008,in);
	hipMemcpy(image,in,sizeof(char)*imagesize,hipMemcpyDeviceToHost);

	write_ppm(image,256,256,"julia.ppm");
	free(image);
	return 0;
}


